﻿
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#define SIZE 10
 
__global__ void test(int *a, int *b){
    int i = threadIdx.x;
    b[i] = a[i] + 1;
}
 
int main(){
    int *a, *b;
    int *d_a, *d_b; 
 
    a = (int *)malloc(SIZE*sizeof(int));
    b = (int *)malloc(SIZE*sizeof(int));


    hipMalloc(&d_a, SIZE*sizeof(int));
    hipMalloc(&d_b, SIZE*sizeof(int));
    
    for (int i = 0; i<SIZE; ++i)
    {
        a[i] = i;
        b[i] = 0;
    }
    
    
    hipMemcpy(d_a, a, SIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, SIZE*sizeof(int), hipMemcpyHostToDevice);
    test <<< 1, SIZE >>>(d_a, d_b); // launch test function
    hipMemcpy(b, d_b, SIZE*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i<SIZE; i++)
        printf("b[%d] = %d\n", i, b[i]);    // print the results
 
    free(a);    // free the host memory spaces
    free(b);    // free the host memory spaces
    
    hipFree(d_a);    // free the device memory spaces 
    hipFree(d_b);    // free the device memory spaces 
    return 0;
}